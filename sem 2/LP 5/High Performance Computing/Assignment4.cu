
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;
using namespace std::chrono;

__global__ void vectorAddCUDA(int *a, int *b, int *c, int n)
{
    //a[],b[],c[] are arrays which are passed in function
    //every thread has unique index in gpu. dim->no. of thread per block, 
    //idx->id of block, threadidx->id of thread inside block 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
    { //if index is in range add elements of a and b and store in c
        c[idx] = a[idx] + b[idx];
    }
}

void vectorAddCPU(int *a, int *b, int *c ,int n)
{
    for (int i = 0; i < n; i++)
    { //runs on cpu
        c[i] = a[i]+ b[i];
        //for (int i=0;i<n;i++)
        //cout<<c[i]<< " ";
        //cout<<endl;
    }
}

__global__ void matrixMulCUDA(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    { 
        int sum=0;
        for(int k=0; k<n; k++)
        sum += a[row*n+k] * b[k*n +col];
        c[row*n + col]= sum;
    }
}

void matrixMulCPU(int *a, int*b, int* c, int n)
{
    for (int i = 0; i < n; i++)
    {
        for(int j=0; j<n; j++)
        {
            int sum =0;
            for (int k =0;k<n;k++)
            sum += a[i*n+k] * b[k*n+j];
            c[i*n+j]=sum;
        }
    }
}

int main()
{
    int vecSize = 1<< 24;   // 16 million (=2^24)
    int matrixSize = 1024; //1024x1024 matrix , 10^5

    int  *h_a = new int [vecSize]; //alocates memory on host 'a' by creating vector
    int  *h_b = new int [vecSize];
    int  *h_c_cpu = new int [vecSize]; 
    int  *h_c_gpu = new int [vecSize];

    for(int i=0; i<vecSize; i++)
    { // fills a and b with random nos b/w 0 and 99
        h_a[i]= rand() % 100;
        h_b[i]= rand() % 100;
    }

    auto start= high_resolution_clock::now();
    vectorAddCPU(h_a, h_b, h_c_cpu, vecSize);
    auto end= high_resolution_clock::now();
    duration<double> vaddcputime = end - start;
    cout<< "Vector Addition[CPU] time:" <<vaddcputime.count()<<"seconds"<<endl;

    //cudamalloc allocates memory on gpu
    //cudamemcpy copies data from cpu to gpu

    int *d_a , *d_b, *d_c; //device
    hipMalloc(&d_a, vecSize * sizeof(int)); 
    hipMalloc(&d_b, vecSize * sizeof(int));
    hipMalloc(&d_c, vecSize * sizeof(int));

    hipMemcpy(d_a, h_a, vecSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vecSize * sizeof(int), hipMemcpyHostToDevice);

    start = high_resolution_clock::now();
    vectorAddCUDA<<<(vecSize + 255)/256, 256>>>(d_a,d_b,d_c,vecSize); //performing addition on gpu
    hipDeviceSynchronize();
    end= high_resolution_clock::now();
    duration<double> vaddgputime = end - start;
    hipMemcpy(h_c_gpu, d_c, vecSize * sizeof(int), hipMemcpyDeviceToHost);
    cout<< "Vector Addition[GPU] Time: "<<vaddgputime.count()<<"seconds"<<endl; 

    int *mat_a = new int[matrixSize * matrixSize]; //making matrix
    int *mat_b = new int[matrixSize * matrixSize];
    int *mat_c_cpu = new int[matrixSize * matrixSize];
    int *mat_c_gpu = new int[matrixSize * matrixSize];

    for(int i=0;i<matrixSize*matrixSize;i++)
    {
        mat_a[i]=rand()%100;
        mat_b[i]=rand()%100;
    }

    start=high_resolution_clock::now();
    matrixMulCPU(mat_a, mat_b, mat_c_cpu, matrixSize);
    end = high_resolution_clock::now();
    duration<double> mmulcpu = end -start;
    cout<< "Matrix Multiplication[CPU] Time: "<<mmulcpu.count()<<"seconds"<<endl;

    int *d_mata, *d_matb, *d_matc;
    hipMalloc(&d_mata, vecSize * sizeof(int));
    hipMalloc(&d_matb, vecSize * sizeof(int));
    hipMalloc(&d_matc, vecSize * sizeof(int));
    hipMemcpy(d_mata,mat_a,matrixSize*matrixSize*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_matb,mat_b,matrixSize*matrixSize*sizeof(int),hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16,16); //2D grid of blocks, 16X16 threads each block
    dim3 blocksPerGrid((matrixSize+15)/16, (matrixSize+15)/16);

    start= high_resolution_clock::now();
    matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_mata, d_matb, d_matc, matrixSize);
    hipDeviceSynchronize();
    end= high_resolution_clock::now();
    duration <double> mmulgpu = end-start;
    cout<<"Matrix Multiplication[GPU] Time: "<<mmulgpu.count()<<"seconds"<<endl;

    delete[] h_a;
    delete[] h_b;
    delete[] h_c_cpu;
    delete[] h_c_gpu;
    delete[] mat_a;
    delete[] mat_b;
    delete[] mat_c_cpu;
    delete[] mat_c_gpu;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_mata);
    hipFree(d_matb);
    hipFree(d_matc);

    return 0;
}